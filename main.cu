#include<stdio.h>
#include<hip/hip_runtime.h>
#include"ghost.h"
#include"complex.h"

int main(){
	typedef unsigned int	Size;
	typedef float			Decimal;

	Size size=1000;
	ghost::ghostRender<Decimal,Size> renderer(size);

	renderer.setConstant(.285,.01);
	renderer.setCenter(1,-1);
	renderer.setRange(2,2);

	renderer.calculateJulia();
	renderer.renderJuliaImage("julia.bmp"); 

	return 0;
}
